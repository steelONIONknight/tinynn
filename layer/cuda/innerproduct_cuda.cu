#include "hip/hip_runtime.h"
//
// Created by lifan on 2021/3/26.
//
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "../../cuda_util.h"

#include <iostream>

#include "../../mat.h"
#include "../innerproduct.h"

const int N_ITERATIONS = 32;

__global__ void cuda_innerproduct_forward(const float* input, const float* weight, float* output,
                                          const int w, const int h, const int c, const int num_output)
{
    unsigned int rowIdx = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int colIdx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int chaIdx = blockIdx.z * blockDim.z + threadIdx.z;

    unsigned int idx = rowIdx * w + colIdx + chaIdx * w * h;
    unsigned int output_idx;

    for (int p = 0; p < num_output; p++)
    {
        output_idx = idx + p * w * h * c;
        output[output_idx] = input[idx] * weight[output_idx];
    }

}

__global__ void cuda_innerproduct_reduction(float* input, float* output, unsigned int n)
{
    unsigned int tid = threadIdx.x;
    //unroll loop 8
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    float* i_data = input + blockIdx.x * blockDim.x * 8;

    if (tid + blockDim.x * 7 < n)
    {
        float a1 = input[idx + 0 * blockDim.x];
        float a2 = input[idx + 1 * blockDim.x];
        float a3 = input[idx + 2 * blockDim.x];
        float a4 = input[idx + 3 * blockDim.x];
        float b1 = input[idx + 4 * blockDim.x];
        float b2 = input[idx + 5 * blockDim.x];
        float b3 = input[idx + 6 * blockDim.x];
        float b4 = input[idx + 7 * blockDim.x];
        input[idx] = a1 + a2 + a3 + a4 + b1 + b2 + b3 + b4;

        __syncthreads();
    }

    for (int stride = blockDim.x / 2; stride > 32; stride /= 2)
    {
        if (tid < stride)
        {
            i_data[tid] += i_data[tid + stride];
        }
        __syncthreads();
    }

    if (tid < 32)
    {
        volatile float* vmem = i_data;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid + 8];
        vmem[tid] += vmem[tid + 4];
        vmem[tid] += vmem[tid + 2];
        vmem[tid] += vmem[tid + 1];
    }

    if (tid == 0)
    {
        output[blockIdx.x] = i_data[0];
    }
}

__global__ void cuda_innerproduct_activation(const float* input, const float* bias_data, const float* activation_params,
                                             float* output, int activation_type, int num_output)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx > num_output)
        return;

    output[idx] = input[idx] + bias_data[idx];

    if (activation_type == 1)
    {
        output[idx] = max(output[idx], 0.f);
    }
    else if (activation_type == 2)
    {
        float slope = activation_params[0];
        output[idx] = output[idx] > 0.f ? output[idx] : output[idx] * slope;
    }
    else if (activation_type == 3)
    {
        float min_val = activation_params[0];
        float max_val = activation_params[1];

        if (output[idx] < min_val)
            output[idx] = min_val;
        if (output[idx] > max_val)
            output[idx] = max_val;
    }
    else if (activation_type == 4)
    {
        output[idx] = (float)(1.f / 1.f + expf(-output[idx]));
    }
    else if (activation_type == 5)
    {
        output[idx] = (float)(output[idx] * tanhf(logf(expf(output[idx]) + 1.f)));
    }

}

__global__ void cuda_innerproduct_reduction_activation(const float* input, float* output, const float* bias_data,
                                                       const float* activation_params, int width, int activation_type)
{
    unsigned int laneIdx = threadIdx.x & 31;
    unsigned int colIdx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int rowIdx = N_ITERATIONS * blockIdx.y;

    unsigned int idx = rowIdx * width + colIdx;

    for (int i = 0; i < N_ITERATIONS; i++)
    {
        float v = idx < width * (i + 1) ? input[idx] : 0.f;

        for (int stride = 16; stride >= 1; stride /= 2)
            v += __shfl_down_sync(0xffffffff, v, stride);

        if (laneIdx == 0)
            atomicAdd(&output[rowIdx], v);

        __syncthreads();

        if (blockIdx.x == 0 && threadIdx.x == 0)
        {
            if (bias_data != nullptr)
                output[rowIdx] += bias_data[rowIdx];
            if (activation_type == 1)
            {
                output[rowIdx] = max(output[rowIdx], 0.f);
            }
            else if (activation_type == 2)
            {
                float slope = activation_params[0];
                output[rowIdx] = output[rowIdx] > 0.f ? output[rowIdx] : output[rowIdx] * slope;
            }
            else if (activation_type == 3)
            {
                float min_val = activation_params[0];
                float max_val = activation_params[1];

                if (output[rowIdx] < min_val)
                    output[rowIdx] = min_val;
                if (output[rowIdx] > max_val)
                    output[rowIdx] = max_val;
            }
            else if (activation_type == 4)
            {
                output[rowIdx] = (float)(1.f / 1.f + expf(-output[rowIdx]));
            }
            else if (activation_type == 5)
            {
                output[rowIdx] = (float)(output[rowIdx] * tanhf(logf(expf(output[rowIdx]) + 1.f)));
            }
        }
        rowIdx++;
        idx += width;
    }
}
namespace tinynn
{
int innerproduct_forward(const CudaMat& bottom_blob, const CudaMat& weight, const CudaMat& activation_params,
                         const CudaMat& bias_data, CudaMat& top_blob, int num_output, int bias_term, int weight_data_size,
                         int activation_type)
{
    dim3 block;
    dim3 grid;
    int dims = bottom_blob.dims;
    int block_x = 1, block_y = 1, block_z = 1;
    int grid_x = 1, grid_y = 1, grid_z = 1;
    if (dims == 1)
    {
        block_x = ((bottom_blob.width - 1) / 32 + 1) * 32;
        block_x = block_x > 1024 ? 1024 : block_x;

        grid_x = (bottom_blob.width - 1) / block_x + 1;

        block.x = block_x;
        block.y = block_y;
        block.z = block_z;
        grid.x = grid_x;
        grid.y = grid_y;
        grid.z = grid_z;
    }
    else if (dims == 2)
    {
        block_x = ((bottom_blob.width - 1) / 32 + 1) * 32;
        block_x = block_x > 1024 ? 1024 : block_x;
        block_y = ((bottom_blob.height - 1) / 32 + 1) * 32;
        block_y = block_y > 1024 ? 1024 : block_y;

        grid_x = (bottom_blob.width - 1) / block_x + 1;
        grid_y = (bottom_blob.height - 1) / block_y + 1;

        block.x = block_x;
        block.y = block_y;
        block.z = block_z;
        grid.x = grid_x;
        grid.y = grid_y;
        grid.z = grid_z;
    }
    else if (dims == 3)
    {
        block_x = ((bottom_blob.width - 1) / 32 + 1) * 32;
        block_x = block_x > 1024 ? 1024 : block_x;
        block_y = ((bottom_blob.height - 1) / 32 + 1) * 32;
        block_y = block_y > 1024 ? 1024 : block_y;

        grid_x = (bottom_blob.width - 1) / block_x + 1;
        grid_y = (bottom_blob.height - 1) / block_y + 1;
        grid_z = (bottom_blob.channel - 1) /block_z + 1;

        block.x = block_x;
        block.y = block_y;
        block.z = block_z;
        grid.x = grid_x;
        grid.y = grid_y;
        grid.z = grid_z;
    }

    float* intermediate_res;
    hipMalloc((void**)& intermediate_res, weight_data_size * sizeof(float));

    cuda_innerproduct_forward<<<grid, block>>>((float*)bottom_blob.data, (float*)weight.data, intermediate_res,
                                               bottom_blob.width, bottom_blob.height, bottom_blob.channel, num_output);
    hipDeviceSynchronize();

    //**************just for debug***************************
//    float* h_intermediate_res;
//    h_intermediate_res = (float*)malloc(weight_data_size * sizeof(float));
//    hipMemcpy(h_intermediate_res, intermediate_res, weight_data_size * sizeof(float), hipMemcpyDeviceToHost);
//    for (int j = 0; j < weight.channel; j++)
//    {
//        setbuf(stdout, nullptr);
//        printf("channel: %d\n", j);
//        int index = j * weight.width * weight.height;
//        for (int i = 0; i < weight.width * weight.height; i++)
//        {
//            setbuf(stdout, nullptr);
//            printf("%f ", h_intermediate_res[index + i]);
//        }
//        setbuf(stdout, nullptr);
//        printf("\n");
//    }
//    free(h_intermediate_res);
    //**************just for debug***************************

    int sz = weight_data_size / num_output;
    block_x = ((sz - 1) / N_ITERATIONS + 1) * N_ITERATIONS;
    block_x = block_x > 1024 ? 1204 : block_x;
    grid_x = sz / N_ITERATIONS;
    grid_x = grid_x >= 1 ? grid_x : 1;
    grid_y = num_output / N_ITERATIONS;
    grid_y = grid_y >= 1 ? grid_y : 1;

    block.x = block_x;
    block.y = 1;
    block.z = 1;
    grid.x = grid_x;
    grid.y = grid_y;
    grid.z = 1;

    if (bias_term == 0)
    {
        cuda_innerproduct_reduction_activation<<<grid, block>>>(intermediate_res, (float*)top_blob.data,
                                                                nullptr, (float*)activation_params.data,
                                                                bottom_blob.width, activation_type);
    }
    else if (bias_term == 1)
    {
        cuda_innerproduct_reduction_activation<<<grid, block>>>(intermediate_res, (float*)top_blob.data,
                                                                (float*)bias_data.data, (float*)activation_params.data,
                                                                bottom_blob.width, activation_type);
    }
    hipDeviceSynchronize();

    //**************just for debug***************************
//    float* h_top_blob;
//    h_top_blob = (float*)malloc(num_output * sizeof(float));
//    hipMemcpy(h_top_blob, top_blob.data, num_output * sizeof(float), hipMemcpyDeviceToHost);
//    for (int i = 0; i < num_output; i++)
//    {
//        printf("%f ", h_top_blob[i]);
//    }
//    printf("\n");
//    free(h_top_blob);
    //**************just for debug***************************




    //    float* dev_temp;
//    hipMalloc((void**)& dev_temp, grid.x * sizeof(float) * num_output);
//    //gpu内存初始化
//    hipMemset(dev_temp, 0, grid.x * sizeof(float) * num_output);
//    float* temp;
//    temp = (float*)malloc(grid.x * sizeof(float) * num_output);
//
//    hipDeviceSynchronize();
//    hipMemcpy(temp, dev_temp, grid.x * sizeof(float) * num_output, hipMemcpyDeviceToHost);


//    float* sum;
//    sum = (float*)malloc(num_output * sizeof(float));
//    std::fill_n(sum, num_output, 0.f);
//
//    for (int i = 0; i < num_output; i++)
//    {
//        for (int j = 0; j < grid.x; j++)
//        {
//            sum[i] += temp[i * grid.x + j];
//        }
//    }

//    float* dev_sum;
//    hipMalloc((void**)& dev_sum, num_output * sizeof(float));
//    hipMemcpy(dev_sum, sum, num_output * sizeof(float), hipMemcpyHostToDevice);
//
//    block_x = ((num_output - 1) / 32 + 1) * 32;
//    block_x = block_x > 1024 ? 1024 : block_x;
//    grid_x = (num_output - 1) / block_x + 1;
//
//    block.x = block_x;
//    block.y = 1;
//    block.z = 1;
//    grid.x = grid_x;
//    grid.y = 1;
//    grid.z = 1;
//
//    cuda_innerproduct_activation<<<grid, block>>>(dev_sum, (float*)bias_data.data, (float*)activation_params.data,
//                                                  (float*)top_blob.data, activation_type, num_output);
//    hipDeviceSynchronize();

    hipFree(intermediate_res);
//    hipFree(dev_temp);
//    hipFree(dev_sum);
//    free(temp);
//    free(sum);
    return 0;
}

} // namespace tinynn